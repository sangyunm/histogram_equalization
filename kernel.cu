#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "CUDA_equalization.h"

__global__ void calculate_Min_Max(unsigned char* Image, int channels, int* min, int* max);
__global__ void histogram_equalization(unsigned char* Image, int channels, int* min, int* max);
__device__ int New_Pixel_Value(int value, int min, int max);

void Histogram_equalization_cuda(unsigned char* Image, int Height, int Width, int channels) {
	unsigned char* Dev_image = NULL;
	int* Dev_min = NULL;
	int* Dev_max = NULL;

	hipMalloc((void**)&Dev_image, Height * Width * channels); 
	hipMalloc((void**)&Dev_min, channels * sizeof(int));
	hipMalloc((void**)&Dev_max, channels * sizeof(int));

	int min[3] = { 255,255,255 };
	int max[3] = { 0,0,0 };

	hipMemcpy(Dev_image, Image, Height * Width * channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_min, min, sizeof(int) * channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_max, max, sizeof(int) * channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	calculate_Min_Max << <Grid_Image, 1 >> > (Dev_image, channels, Dev_min, Dev_max);
	histogram_equalization << <Grid_Image, 1 >> > (Dev_image, channels, Dev_min, Dev_max);

	hipMemcpy(Image, Dev_image, Height * Width * channels, hipMemcpyDeviceToHost);
	 
	hipFree(Dev_image);
}

__global__ void calculate_Min_Max(unsigned char* Image, int channels, int* min, int* max){
	int x = blockIdx.x;
	int y = blockIdx.y;

	int Image_idx = (x + y * gridDim.x)*channels;
	 
	for (int i = 0; i < channels; i++) {
		atomicMin(&min[i], Image[Image_idx + i]);
		atomicMax(&max[i], Image[Image_idx + i]);
	}
}
__global__ void  histogram_equalization(unsigned char* Image, int channels, int* min, int* max) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int Image_idx = (x + y * gridDim.x) * channels;
	for (int i = 0; i < channels; i++) {
		Image[Image_idx + i] = New_Pixel_Value(Image[Image_idx+i], min[i], max[i]);
	}
}

__device__ int New_Pixel_Value(int value, int min, int max) {
	int target_min = 0;
	int target_max = 255;

	return (target_min + (value - min) * (int)((target_max - target_min) / (max - min)));
}
